#include "hip/hip_runtime.h"
/*
  This file contains the gpu version of the DNN functions
 */
#include "data_types.hpp"
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>
#include <thrust/copy.h>

#define LW 0 // left wall
#define BW 1 // bottom wall
#define RW 2 // right wall
#define TW 3 // top wall

#define CONST 0
#define FLUX 1

#define BLOCK_SIZE 128

__constant__ double gridFo;
__constant__ double d;
__constant__ double dx;
__constant__ double dy;
__constant__ double dt;
__constant__ unsigned int xsize;
__constant__ unsigned int ysize;
__constant__ int bc_type[4];
__constant__ double bc_values[4];
__constant__ int A;
__constant__ int B;
__constant__ int C;

__device__ __forceinline__ int index(int x, int y) { return x + y * xsize; }

__device__ void destroy_needle(int *is_needle, Needle needle) {
  for (int i = needle.x0; i <= needle.xf; ++i) {
    /* int idx = i + needle.y0*xsize; */
    /* int idx = index(i, needle.y0); */

    is_needle[index(i, needle.y0)] = 0;

    int wd = (int)sqrt(2 * needle.rad * (needle.xf - i) * dx) / dy;
    /* wd = wd < 3 ? wd : 3; */
    for (int j = 1; j <= wd; ++j) {
      /* is_needle[idx + j*xsize] = 0; */
      /* is_needle[idx - j*xsize] = 0; */
      is_needle[index(i, needle.y0 + j)] = 0;
      is_needle[index(i, needle.y0 - j)] = 0;
    }
  }
}

__device__ double calculate_line_integral(double *grid, int xb, int xf, int yb,
                                          int yf, int y_nb, int y_nt) {
  double dxu;
  double dyu;
  double flux_acc = 0.0;
  double xfactor = 1.0 / (2.0 * dx);
  double yfactor = 1.0 / (2.0 * dy);

  // Bottom contour line
  // Edge case bottom contour
  dyu = yfactor * (grid[index(xb - 1, yb)] + grid[index(xb, yb)] -
                   grid[index(xb - 1, yb - 1)] - grid[index(xb, yb - 1)]);
  flux_acc -= 0.5 * dyu * dx;
  for (int i = xb; i < xf; ++i) {
    dyu = yfactor * (grid[index(i, yb)] + grid[index(i + 1, yb)] -
                     grid[index(i, yb - 1)] - grid[index(i + 1, yb - 1)]);
    flux_acc -= dyu * dx;
  }
  dyu = yfactor * (grid[index(xf, yb)] + grid[index(xf + 1, yb)] -
                   grid[index(xf, yb - 1)] - grid[index(xf + 1, yb - 1)]);
  flux_acc -= 0.5 * dyu * dx;

  // Front contour line
  dxu = xfactor * (grid[index(xf + 1, yb - 1)] + grid[index(xf + 1, yb)] -
                   grid[index(xf, yb - 1)] - grid[index(xf, yb)]);
  flux_acc += 0.5 * dxu * dy;
  for (int i = yb; i < yf; ++i) {
    dxu = xfactor * (grid[index(xf + 1, i)] + grid[index(xf + 1, i + 1)] -
                     grid[index(xf, i)] - grid[index(xf, i + 1)]);
    flux_acc += dxu * dy;
  }
  dxu = xfactor * (grid[index(xf + 1, yf)] + grid[index(xf + 1, yf + 1)] -
                   grid[index(xf, yf)] - grid[index(xf, yf + 1)]);
  flux_acc += 0.5 * dxu * dy;

  // Top contour line
  dyu = yfactor * (grid[index(xb - 1, yf + 1)] + grid[index(xb, yf + 1)] -
                   grid[index(xb - 1, yf)] - grid[index(xb, yf)]);
  flux_acc += 0.5 * dyu * dx;
  for (int i = xb; i < xf; ++i) {
    dyu = yfactor * (grid[index(i, yf + 1)] + grid[index(i + 1, yf + 1)] -
                     grid[index(i, yf)] - grid[index(i + 1, yf)]);
    flux_acc += dyu * dx;
  }
  dyu = yfactor * (grid[index(xf, yf + 1)] + grid[index(xf + 1, yf + 1)] -
                   grid[index(xf, yf)] - grid[index(xf + 1, yf)]);
  flux_acc += 0.5 * dyu * dx;

  // Back contour line
  // Split in two, due to do needle
  for (int i = y_nt; i < yf; ++i) {
    dxu = xfactor * (grid[index(xb, i)] + grid[index(xb, i + 1)] -
                     grid[index(xb - 1, i)] - grid[index(xb - 1, i + 1)]);
    flux_acc -= dxu * dy;
  }
  dxu = xfactor * (grid[index(xb, yf)] + grid[index(xb, yf + 1)] -
                   grid[index(xb - 1, yf)] - grid[index(xb - 1, yf + 1)]);
  flux_acc -= 0.5 * dxu * dy;

  dxu = xfactor * (grid[index(xb, yb - 1)] + grid[index(xb, yb)] -
                   grid[index(xb - 1, yb - 1)] - grid[index(xb - 1, yb)]);
  flux_acc -= 0.5 * dxu * dy;
  for (int i = yb; i < y_nb; ++i) {
    dxu = xfactor * (grid[index(xb, i)] + grid[index(xb, i + 1)] -
                     grid[index(xb - 1, i)] - grid[index(xb - 1, i + 1)]);
    flux_acc -= dxu * dy;
  }

  return flux_acc;
}

__device__ double calculate_surface_integral(double *grid, int *is_needle,
                                             int xb, int xf, int yb, int yf) {
  double xfactor = 1.0 / (2.0 * dx);
  double integral = 0.0;
  double dxu;

  for (int y = yb; y <= yf; ++y) {
    for (int x = xb; x <= xf; ++x) {
      if (is_needle[index(x, y)] == 0) {
        dxu = xfactor * (grid[index(x + 1, y)] - grid[index(x - 1, y)]);
        integral += dxu * dx * dy;
      }
    }
  }

  return integral;
}

__device__ double calculate_flux_intensity_factor(double *grid, int *is_needle,
                                                  Needle needle) {
  double factor = 1 / (4 * sqrt((A + needle.r + 1 / 2) * dx));

  int xb = needle.xf - A;
  int xf = needle.xf + B;

  int y_nt, y_nb;
  for (int i = 0; i < ysize; ++i) {
    if (is_needle[index(xb, needle.y0 + i)] == 0) {
      y_nt = needle.y0 + i - 1;
      y_nb = needle.y0 - i + 1;
      break;
    }
  }

  int yf = y_nt + C;
  int yb = y_nb - C;

  double line = calculate_line_integral(grid, xb, xf, yb, yf, y_nb, y_nt);
  double surface = calculate_surface_integral(grid, is_needle, xb, xf, yb, yf);
  /* printf("\n\nIntegral values line: %f, surface: %f\n\n", line, surface); */
  surface = (needle.vel * surface) / d;

  return (line + surface) * factor;
}

__device__ void grow_needle(double *grid, int *is_needle, Needle *needle) {
  double fif = calculate_flux_intensity_factor(grid, is_needle, *needle);

  /* printf("\nFIF calcuated: %f\n\n", fif); */

  double fif_factor = fif * fif * 2 * d * d;
  double vel = pow(fif_factor, 0.6666666666);
  double rad = pow(fif_factor, -0.3333333333);

  if (rad < needle->rad) {
    destroy_needle(is_needle, *needle);
  }

  /* printf("\nNeedle rad: %f, vel: %f, r: %f\n\n", needle->rad, needle->vel,
   * needle->r); */
  /* printf("\nNew needle rad: %f, vel: %f, r: %f\n\n", vel, rad, 0.0); */

  needle->r += (vel * dt) / dx;
  needle->rad = rad;
  needle->vel = vel;

  if (needle->r > 1.0) {
    needle->xf += 1;
    needle->r -= 1.0;
  }
}

__global__ void print_device_constants() {
  int idx = threadIdx.x;

  if (idx == 0) {
    printf("device_gridFo: %f\n", gridFo);
    printf("device_d: %f\n", d);
    printf("device_dx: %f\n", dx);
    printf("device_dy: %f\n", dy);
    printf("device_xsize: %d\n", xsize);
    printf("device_ysize: %d\n", ysize);

    for (int i = 0; i < 4; ++i) {
      printf("BC_TYPE[%d]: %d, BC_VAL[%d], %f\n", i, bc_type[i], i,
             bc_values[i]);
    }
  }
}

__global__ void print_needles(Needle *needles, int size) {
  int idx = threadIdx.x;

  if (idx == 0) {

    for (int i = 0; i < size; ++i) {
      printf("Device: (x0: %d, y0: %d), (xf: %d, yf: %d), rad: %f, vel: %f, r: "
             "%f\n",
             needles[i].x0, needles[i].y0, needles[i].xf, needles[i].yf,
             needles[i].rad, needles[i].vel, needles[i].r);
    }
  }
}

__global__ void grow_needles(double *grid, int *is_needle, Needle *needles,
                             int size) {
  // Assuming needle is alinged along the x-axis
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx < size) {
    grow_needle(grid, is_needle, &needles[idx]);
  }
}

__device__ void set_needle(double *grid, int *is_needle, Needle needle) {
  for (int i = needle.x0; i <= needle.xf; ++i) {
    /* int idx = i + needle.y0*xsize; */
    int idx = index(i, needle.y0);

    grid[idx] = 0;
    is_needle[idx] = 1;

    int wd = (int)sqrt(2 * needle.rad * (needle.xf - i) * dx) / dy;
    /* wd = wd < 3 ? wd : 3; */
    for (int j = 1; j <= wd; ++j) {
      /* grid[idx + j*xsize] = 0; */
      /* grid[idx - j*xsize] = 0; */
      /* is_needle[idx + j*xsize] = 1; */
      /* is_needle[idx - j*xsize] = 1; */
      int idx_t = index(i, needle.y0 + j);
      int idx_b = index(i, needle.y0 - j);
      grid[idx_t] = 0;
      grid[idx_b] = 0;
      is_needle[idx_t] = 1;
      is_needle[idx_b] = 1;
    }
  }
}

__device__ void shift_row(double *grid, int *is_needle, int start_index) {
  thrust::copy(thrust::device, grid + start_index + 1,
               grid + start_index + xsize, grid + start_index);
  thrust::copy(thrust::device, is_needle + start_index + 1,
               is_needle + start_index + xsize, is_needle + start_index);
}

__global__ void set_needles(double *grid, int *is_needle, Needle *needles,
                            int size) {
  // Assuming needly only aligned along the x-axis
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx < size) {
    set_needle(grid, is_needle, needles[idx]);
  }
}

__global__ void shift_domain(double *grid, int *is_needle, Needle *needles,
                             double pos) {
  if (needles[0].xf > pos) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx < ysize) {
      shift_row(grid, is_needle, idx * xsize);
      if (idx == 0) {
        needles[0].xf = needles[0].xf - 1;
      }
    }
  }
}

__global__ void explicit_euler(double *grid, double *ogrid) {
  // I do not see any reason why the solver shoulde be aware of the needle
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < xsize * ysize) {
    int idxl = idx - 1;
    int idxr = idx + 1;
    int idxt = idx + xsize;
    int idxb = idx - xsize;
    double nval;
    if (idx == 0) {
      // Left Bottom corner
      int bc = bc_type[LW] + bc_type[BW];
      if (bc == 2 * FLUX) {
        nval =
            grid[idx] + gridFo * (2 * grid[idxr] + 2 * dx * bc_values[LW] / d +
                                  2 * grid[idxt] + 2 * dy * bc_values[BW] / d -
                                  4 * grid[idx]);
      } else if (bc == FLUX) {
        if (bc_type[LW] == 0) {
          nval = bc_values[LW];
        } else {
          nval = bc_values[BW];
        }
      } else {
        nval = (bc_values[LW] + bc_values[BW]) / 2;
      }
    } else if (idx == xsize - 1) {
      // Right bottom corner
      int bc = bc_type[RW] + bc_type[BW];
      if (bc == 2 * FLUX) {
        nval =
            grid[idx] + gridFo * (2 * grid[idxl] - 2 * dx * bc_values[RW] / d +
                                  2 * grid[idxt] + 2 * dy * bc_values[BW] / d -
                                  4 * grid[idx]);
      } else if (bc == FLUX) {
        if (bc_type[RW] == 0) {
          nval = bc_values[RW];
        } else {
          nval = bc_values[BW];
        }
      } else {
        nval = (bc_values[RW] + bc_values[BW]) / 2;
      }
    } else if (idx == (ysize - 1) * xsize) {
      // Left top corner
      int bc = bc_type[LW] + bc_type[TW];
      if (bc == 2 * FLUX) {
        nval =
            grid[idx] + gridFo * (2 * grid[idxr] + 2 * dx * bc_values[LW] / d +
                                  2 * grid[idxb] - 2 * dy * bc_values[TW] / d -
                                  4 * grid[idx]);
      } else if (bc == FLUX) {
        if (bc_type[LW] == 0) {
          nval = bc_values[LW];
        } else {
          nval = bc_values[TW];
        }
      } else {
        nval = (bc_values[LW] + bc_values[TW]) / 2;
      }
    } else if (idx == xsize * ysize - 1) {
      // Right top corner
      int bc = bc_type[RW] + bc_type[TW];
      if (bc == 2 * FLUX) {
        nval =
            grid[idx] + gridFo * (2 * grid[idxl] - 2 * dx * bc_values[RW] / d +
                                  2 * grid[idxb] - 2 * dy * bc_values[TW] / d -
                                  4 * grid[idx]);
      } else if (bc == FLUX) {
        if (bc_type[LW] == 0) {
          nval = bc_values[RW];
        } else {
          nval = bc_values[TW];
        }
      } else {
        nval = (bc_values[RW] + bc_values[TW]) / 2;
      }
    } else if (idx < xsize) {
      // Bottom wall
      if (bc_type[BW] == FLUX) {
        nval =
            grid[idx] + gridFo * (grid[idxr] + grid[idxl] + 2 * grid[idxt] +
                                  2 * dy * bc_values[BW] / d - 4 * grid[idx]);
      } else {
        nval = bc_values[BW];
      }
    } else if ((idx + 1) % xsize == 0) {
      // Right wall
      if (bc_type[RW] == FLUX) {
        nval =
            grid[idx] + gridFo * (2 * grid[idxl] - 2 * dx * bc_values[RW] / d +
                                  grid[idxt] + grid[idxb] - 4 * grid[idx]);
      } else {
        nval = bc_values[RW];
      }
    } else if (idx % xsize == 0) {
      // Left Wall
      if (bc_type[LW] == FLUX) {
        nval =
            grid[idx] + gridFo * (2 * grid[idxr] + 2 * dx * bc_values[LW] / d +
                                  grid[idxt] + grid[idxb] - 4 * grid[idx]);
      } else {
        nval = bc_values[LW];
      }
    } else if (idx > (ysize - 1) * xsize) {
      // Top wall
      if (bc_type[TW] == FLUX) {
        nval =
            grid[idx] + gridFo * (grid[idxr] + grid[idxl] + 2 * grid[idxb] -
                                  2 * dy * bc_values[TW] / d - 4 * grid[idx]);
      } else {
        nval = bc_values[TW];
      }
    } else {
      // Inner domain
      nval = grid[idx] + gridFo * (grid[idxl] + grid[idxr] + grid[idxt] +
                                   grid[idxb] - 4 * grid[idx]);
    }
    ogrid[idx] = nval;
  }
}

void run_transient_CUDA(double *d_grid, double *d_ogrid, int size) {
  int nthreads = BLOCK_SIZE;
  int nblocks = ceil((double)size / (double)nthreads);
  explicit_euler<<<nblocks, nthreads>>>(d_grid, d_ogrid);
}

void set_parabolic_needle(double *d_grid, int *d_gneedle, Needle *d_needles,
                          int size) {
  int block_size = 32;
  int nblocks =
      ceil(static_cast<double>(size) / static_cast<double>(block_size));
  set_needles<<<nblocks, block_size>>>(d_grid, d_gneedle, d_needles, size);
  /* print_needles<<<1, 32>>>(needles, size); */
  /* hipDeviceSynchronize(); */
}

void run_grow_needles(double *d_grid, int *d_gneedle, Needle *d_needles,
                      int size) {
  int block_size = 32;
  int nblocks =
      ceil(static_cast<double>(size) / static_cast<double>(block_size));
  grow_needles<<<nblocks, block_size>>>(d_grid, d_gneedle, d_needles, size);
}

void run_shift_domain(double *d_grid, int *d_gneedle, Needle *d_needles,
                      int ysize, double pos) {
  int block_size = 32;
  int nblocks =
      ceil(static_cast<double>(ysize) / static_cast<double>(block_size));
  shift_domain<<<nblocks, block_size>>>(d_grid, d_gneedle, d_needles, pos);
}

void init_CUDA(double _gridFo, double _d, double _dx, double _dy, double _dt,
               unsigned int _xsize, unsigned int _ysize, int _A, int _B, int _C,
               int _bc_type[4], double _bc_values[4]) {
  printf("INIT_CUDA Function\n");
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gridFo), &_gridFo, sizeof(double)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d), &_d, sizeof(double)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dx), &_dx, sizeof(double)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dy), &_dy, sizeof(double)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dt), &_dt, sizeof(double)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(xsize), &_xsize, sizeof(unsigned int)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ysize), &_ysize, sizeof(unsigned int)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(A), &_A, sizeof(int)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(B), &_B, sizeof(int)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(C), &_C, sizeof(int)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(bc_type), _bc_type, 4 * sizeof(int)));
  checkCudaErrors(
      hipMemcpyToSymbol(HIP_SYMBOL(bc_values), _bc_values, 4 * sizeof(double)));
  /* print_device_constants<<<1, 32>>>(); */
  /* hipDeviceSynchronize(); */
}

void copy_to_device_memory(void *d_ptr, void *h_ptr, int bytes) {
  /* printf("\n\nCopy to device memory called.\n\n"); */
  checkCudaErrors(hipMemcpy(d_ptr, h_ptr, bytes, hipMemcpyHostToDevice));
}

void copy_to_host_memory(void *h_ptr, void *d_ptr, int bytes) {
  /* printf("\n\nCopy to host memory called.\n\n"); */
  checkCudaErrors(hipMemcpy(h_ptr, d_ptr, bytes, hipMemcpyDeviceToHost));
}

void allocate_device_memory(void **d_ptr_ptr, int bytes) {
  /* printf("\n\nAllocate to device memory called.\n\n"); */
  checkCudaErrors(hipMalloc(d_ptr_ptr, bytes));
}

void cuda_synchronize_device() { checkCudaErrors(hipDeviceSynchronize()); }

void cuda_release_memory(void *d_ptr) { checkCudaErrors(hipFree(d_ptr)); }
